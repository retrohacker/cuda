
#include <hip/hip_runtime.h>
#include <stdio.h>
#define M 3
#define N 3
#define P 3

__global__ void kernel(float*,float*,float*);
void random_floats(float*,int);
void print_matrix(float*,int,int);

int main(int argc,char** argv) {
  /**
   * Init all variables
   */
  int a_size      = sizeof(float)*M*N,
      b_size      = sizeof(float)*N*P,
      result_size = sizeof(float)*M*P;

  float a[]        = {1,2,3,4,5,6,7,8,9},
        b[]        = {9,8,7,6,5,4,3,2,1},
        answer[]   = {30,24,18,84,69,54,138,114,90},
        *result    = (float*)malloc(result_size),
        *d_a,
        *d_b,
        *d_result;

  /**
   * Setup device memory
   */
  hipMalloc((void**)&d_a,a_size);
  hipMalloc((void**)&d_b,b_size);
  hipMalloc((void**)&d_result,result_size);
  hipMemcpy(d_a,a,a_size,hipMemcpyHostToDevice);
  hipMemcpy(d_b,b,b_size,hipMemcpyHostToDevice);

  /**
   * Start GPU
   */
  kernel<<<P,M>>>(d_a,d_b,d_result);

  /**
   * Copy results back to host
   */
  hipMemcpy(result,d_result,sizeof(float)* M * P,hipMemcpyDeviceToHost);

  /**
   * Print results
   */
  printf("Result: \n");
  print_matrix(result,M,P);
  printf("Expected: \n");
  print_matrix(answer,M,P);

  /**
   * Cleanup memory
   */
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_result);
  free(result);
  return 0;
}

void print_matrix(float *a,int cols,int rows) {
  int i,j;
  for(i=0;i<cols;i++) {
    for(j=0;j<rows;j++)
      printf("%f ",a[i*M+j]);
    printf("\n");
  }
}

__global__ void kernel(float *a,float *b,float *result) {
  bool extra_a;
  int row = blockIdx.x,
      col = threadIdx.x,
      a_count,
      offset,
      i;

  /**
   * Allocate shared memory
   */
  __shared__ float local_a[M];
  __shared__ float local_b[N*P];

  /**
   * Each thread is responsible for loading:
   * 1. An entire column from table b
   * 2. Thread 0 loads row from a
   */
  extra_a = M%blockDim.x>0&&M%blockDim.x<threadIdx.x;
  a_count = (extra_a)?M/blockDim.x+1:M/blockDim.x;
  offset  = (extra_a)?a_count*threadIdx.x:a_count*threadIdx.x+M%blockDim.x;
  for(i=0;i<a_count;i++)
    local_a[offset+i] = a[row*M+offset+i];

  for(i=0;i<P;i++) {
    offset = i*N+threadIdx.x;
    local_b[offset] = b[offset];
  }
  __syncthreads();

  /**
   * Computer cell value
   */
  for(result[row*M+col]=0,i=0;i<N;i++)
    result[row*M+col] += local_a[i] * local_b[i*N+col];
}

void random_floats(float* a, int size) {
  int i;
  for(i=0;i<size;i++)
    a[i] = rand() % 8 + 1; //generate a number betwee 1 and 9
}
